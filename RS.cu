//MatrixMult.cu
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>

__global__ void gpu_sort(int *a,int *b,int *c, int N) {
	int tid = threadIdx.x + blockDim.x * blockIdx.x; 
	int count = 0;
	int d;
		for(d=0;d<N;d++) {
			if(a[d] < a[tid]) {
				count++;
			}
		}
		c[count] = a[tid];
	
}


int main(int argc, char *argv[])  {
	int i, j; 							// loop counters
	int Grid_Dim_x=1, Grid_Dim_y=1;		//Grid structure values
	int Block_Dim_x=1, Block_Dim_y=1;		//Block structure values
	int noThreads_x, noThreads_y;			// number of threads available in device, each dimension
	int noThreads_block;					// number of threads in a block
	int N = 10;  						// size of array in each dimension
	int B;
	int T;
	int *a,*b,*c,*d;
	int *dev_a, *dev_b, *dev_c;
	int size;							// number of bytes in arrays
	hipEvent_t start, stop;     				// using cuda events to measure time
	float elapsed_time_ms;       			// which is applicable for asynchronous code also
	hipEventCreate(&start);		
	hipEventCreate(&stop);


/* --------------------ENTER INPUT PARAMETERS AND ALLOCATE DATA -----------------------*/
							// keyboard input

	printf("Enter the value for N: ");
	scanf("%d", &N);
//takes in input
	int valid = 0;
	while(valid == 0) {

		printf("Enter the number of blocks: ");
		scanf("%d", &B);

		printf("Enter the number of threads: ");
		scanf("%d", &T);

		if(B > 1024 || T > 1024 || B*T < N) {
			printf("Invlaid input entered.\n");
		} else {
			valid = 1;
			Grid_Dim_x = B;
			Block_Dim_x = T;		//puts the size of blocks and thread in for the dim3
		}
	}


	
	dim3 Grid(Grid_Dim_x, Grid_Dim_x);	//Grid structure
	dim3 Block(Block_Dim_x,Block_Dim_y);	//Block structure, threads/block limited by specific device
	size = N * N * sizeof(int);				// number of bytes in total in arrays

	a = (int*) malloc(size);					//dynamically allocated memory for arrays on host
	b = (int*) malloc(size);
	c = (int*) malloc(size);					// results from GPU
	d = (int*) malloc(size);				// results from CPU
							// load arrays with some numbers

	srand(3); //initialize random number generator
	
	for (i=0; i < N; i++) { //load array with numbers
		a[i] = (int)rand(); 
	}


	hipMalloc((void**)&dev_a, size);			// allocate memory on device
	hipMalloc((void**)&dev_b, size);
	hipMalloc((void**)&dev_c, size);

	hipMemcpy(dev_a, a , size ,hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b , size ,hipMemcpyHostToDevice);

	hipEventRecord(start, 0); 			// here start time, after memcpy

	gpu_sort<<<Grid,Block>>>(dev_a,dev_b,dev_c,N);
	hipMemcpy(c, dev_c, size , hipMemcpyDeviceToHost);

	hipEventRecord(stop, 0);     			// measuse end time
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed_time_ms, start, stop );

	printf("Time to calculate results on GPU: %f ms.\n", elapsed_time_ms);
	double gpuTime = elapsed_time_ms; 

/* ------------- COMPUTATION DONE ON HOST CPU ----------------------------*/

	hipEventRecord(start, 0);			// use same timing*

	//cpu_matrixmult(a,b,d,N);				// do calculation on host
//sequential rank sort
	int k;
	for(k=0;k<N;k++) {
		int count = 0;
		int d;
		for(d=0;d<N;d++) {
			if(a[d] < a[k]) {
				count++;
			}
		}
		b[count] = a[k];
		count = 0;
	}


	hipEventRecord(stop, 0);     		// measure end time
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed_time_ms, start, stop );

	printf("Time to calculate results on CPU: %f ms.\n", elapsed_time_ms);  // exe. time
	double cpuTime = elapsed_time_ms;

/* ------------------- check device creates correct results -----------------*/
/*
	printf("Initial Array: \n");
	int h;
	for(h=0;h<N;h++) {
		printf("%d ", a[h]);
	}

	printf("\n");
/*
	printf("Sequential Rank Sort: \n");
	
	for(k=0;k<N;k++) {
		int count = 0;
		int d;
		for(d=0;d<N;d++) {
			if(a[d] < a[k]) {
				count++;
			}
		}
		b[count] = a[k];
		count = 0;
	}

	for(h=0;h<N;h++) {
		printf("%d ", b[h]);
	}

printf("Parallel Rank Sort\n");
	for(h=0;h<N;h++) {
		printf("%d ", c[h]);
	}
*/

int error = 0;
int r;
for(r=0;r<N;r++) {
	if(b[r] != c[r]) {
		error = 1;
		break;
	}
}
if(error == 1) {
	printf("Parallel and sequential do not match.\n");
} else {
	printf("Seqential and parallel match.\n");
}
	printf("Speedup Factor: %lf\n", cpuTime/gpuTime);


/* --------------------- repeat program  ----------------------------------------*/
 								//  while loop to repeat calc with different parameters
/* --------------  clean up  ---------------------------------------*/
	free(a); free(b); free(c);
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	return 0;
}

